#include "hip/hip_runtime.h"
#include "FluidSolver3D.cuh"

//basic funcs################1
FluidSolver3D::FluidSolver3D(int width, int height, int depth, float dx_, float dt_){
    gridWidth = width;
    gridHeight = height;
    gridDepth = depth;
    w_x_h_x_d = gridWidth * gridHeight * gridDepth;
    dx = dx_;
    dy = dx;
    dz = dx;
    dt = dt_;
    std::cout << "dx=" << dx<<"\n";
    h_particles = thrust::host_vector<Utility::Particle3D>();
    d_particles = thrust::device_vector<Utility::Particle3D>();
}

FluidSolver3D::~FluidSolver3D(){
}


__host__ void FluidSolver3D::init(const std::string& fileName) {
    std::ifstream file(fileName);
    if (!file.is_open()) {
        throw std::runtime_error("Cannot open file: " + fileName);
    }

    // Чтение размеров
    file >> gridWidth >> gridHeight >> gridDepth;
    w_x_h_x_d = gridWidth * gridHeight * gridDepth;
    // Инициализация сеток
    labels.resize(gridWidth, gridHeight, gridDepth);
    p.resize(gridWidth, gridHeight, gridDepth);
    u.resize(gridWidth+1, gridHeight, gridDepth);
    v.resize(gridWidth, gridHeight+1, gridDepth);
    w.resize(gridWidth, gridHeight, gridDepth+1);

    uSaved.resize(gridWidth+1, gridHeight, gridDepth);
    vSaved.resize(gridWidth, gridHeight+1, gridDepth);
    wSaved.resize(gridWidth, gridHeight, gridDepth+1);

    thrust::fill(u.device_data.begin(), u.device_data.end(), 0.0f);
    thrust::fill(v.device_data.begin(), v.device_data.end(), 0.0f);
    thrust::fill(w.device_data.begin(), w.device_data.end(), 0.0f);

    thrust::fill(uSaved.device_data.begin(), uSaved.device_data.end(), 0.0f);
    thrust::fill(vSaved.device_data.begin(), vSaved.device_data.end(), 0.0f);
    thrust::fill(wSaved.device_data.begin(), wSaved.device_data.end(), 0.0f);

    // Конфигурация блоков и потоков
    blockSize3D = dim3(8, 8, 8); // 512 потоков в блоке
    threadsPerBlock1D = 256;      // Для обработки частиц

    // Расчет размеров гридов для различных сеток
    gridSizeCells = dim3(
            (gridWidth + blockSize3D.x - 1) / blockSize3D.x,
            (gridHeight + blockSize3D.y - 1) / blockSize3D.y,
            (gridDepth + blockSize3D.z - 1) / blockSize3D.z
    );

    gridSizeU = dim3(
            (gridWidth + 1 + blockSize3D.x - 1) / blockSize3D.x,
            (gridHeight + blockSize3D.y - 1) / blockSize3D.y,
            (gridDepth + blockSize3D.z - 1) / blockSize3D.z
    );

    gridSizeV = dim3(
            (gridWidth + blockSize3D.x - 1) / blockSize3D.x,
            (gridHeight + 1 + blockSize3D.y - 1) / blockSize3D.y,
            (gridDepth + blockSize3D.z - 1) / blockSize3D.z
    );

    gridSizeW = dim3(
            (gridWidth + blockSize3D.x - 1) / blockSize3D.x,
            (gridHeight + blockSize3D.y - 1) / blockSize3D.y,
            (gridDepth + 1 + blockSize3D.z - 1) / blockSize3D.z
    );

    // Чтение меток
    for (int k = gridDepth-1; k > -1; --k) {
        for (int j = gridHeight-1; j > -1; --j) {
            for (int i = gridWidth-1; i > -1; --i) {
                char cellType;
                file >> cellType;

                switch (cellType) {
                    case 'S': labels(i, j, k) = Utility::SOLID; break;
                    case 'F': labels(i, j, k) = Utility::FLUID; break;
                    case 'A': labels(i, j, k) = Utility::AIR;   break;
                }
            }
        }
    }
    labels.copy_to_device();

    // Инициализация частиц
    seedParticles(PARTICLES_PER_CELL);
    std::cout <<"Number of particles is" << h_particles.size() << std::endl;
    Utility::save3dParticlesToPLY(h_particles, "InputData/particles_-1.ply");
}

struct FluidFlagFunctor {
    __host__ __device__
    int operator()(int label) const {
        return label == Utility::FLUID ? 1 : 0;
    }
};

__host__ void FluidSolver3D::seedParticles(int particlesPerCell){
    // Инициализация генератора (один раз вне функции!)
    static std::random_device rd;
    static std::mt19937 gen(rd());
    static std::uniform_int_distribution<> subCellDist(0, 7);
    static std::uniform_real_distribution<> jitterDist(-0.24f, 0.24f);

    // Сначала подсчитываем общее количество частиц
    h_particles.clear();
    size_t totalParticles = 0;
    thrust::device_vector<int> flags(w_x_h_x_d, 0);
    thrust::transform(
            thrust::device,
            labels.device_ptr(),
            labels.device_ptr() + w_x_h_x_d,
            flags.begin(),
            FluidFlagFunctor()
    );
    fluidCellsAmount = thrust::reduce(
            thrust::device,
            flags.begin(),
            flags.end(),
            0,
            thrust::plus<int>()
    );
    totalParticles = fluidCellsAmount * particlesPerCell;

    // Резервируем память заранее
    h_particles.reserve(totalParticles);
    d_particles.reserve(totalParticles);

    // Проходим по всем ячейкам с жидкостью
    for(int k = 0; k < gridDepth; ++k)
        for (int j = 0; j < gridHeight; ++j) {
            for (int i = 0; i < gridWidth; ++i) {
                if (labels(i,j,k) == Utility::FLUID) {
                    float3 cellCenter = Utility::getGridCellPosition(i, j, k, dx);
                    // 8 субрегионов (октантов) в 3D ячейке
                    float3 subCenters[8];
                    const float offset = 0.25f * dx;

                    // Генерируем центры субрегионов
                    for (int octant = 0; octant < 8; ++octant) {
                        subCenters[octant] = {
                                cellCenter.x + (octant & 1 ? offset : -offset),
                                cellCenter.y + (octant & 2 ? offset : -offset),
                                cellCenter.z + (octant & 4 ? offset : -offset)
                        };
                    }

                    // Равномерное распределение частиц по субрегионам
                    for (int pind = 0; pind < particlesPerCell; ++pind) {
                        // Случайный выбор субрегиона для каждой частицы
                        int subCellIdx = subCellDist(gen);

                        // Случайное смещение
                        float jitterX = jitterDist(gen) * dx;
                        float jitterY = jitterDist(gen) * dx;
                        float jitterZ = jitterDist(gen) * dx;

                        // Позиция частицы
                        float3 pos = {
                                subCenters[subCellIdx].x + jitterX,
                                subCenters[subCellIdx].y + jitterY,
                                subCenters[subCellIdx].z + jitterZ
                        };

                        // Ограничение позиции в пределах ячейки
                        pos.x = std::clamp(pos.x, i * dx, (i + 1) * dx);
                        pos.y = std::clamp(pos.y, j * dx, (j + 1) * dx);
                        pos.z = std::clamp(pos.z, k * dx, (k + 1) * dx);

                        // Создаем частицу
                        Utility::Particle3D particle(pos, make_float3(0.0f, 0.0f, 0.0f));

                        // Добавляем в список
                        h_particles.push_back(particle);
                    }
                }
            }
        }
    //thrust::copy(h_particles.begin(), h_particles.end(), d_particles.begin());
    d_particles = h_particles;
    blocksForParticles = (h_particles.size() + threadsPerBlock- 1) / threadsPerBlock;
}


struct ClearNonSolidFunctor {
    __host__ __device__
    int operator()(const int& oldLabel) const {
        return (oldLabel == Utility::SOLID) ? Utility::SOLID : Utility::AIR;
    }
};
struct MarkFluidCellsFunctor {
    const Utility::Particle3D* particles;
    float dx;
    int W, H, D;
    int* labels;    // raw‐pointer на labels.device_data

    MarkFluidCellsFunctor(const Utility::Particle3D* _particles,
                          float _dx, int _W, int _H, int _D,
                          int* _labels)
            : particles(_particles),
              dx(_dx),
              W(_W), H(_H), D(_D),
              labels(_labels) {}

    __device__
    void operator()(int pid) const {
        float3 p = particles[pid].pos;
        int i = static_cast<int>(floorf(p.x / dx));
        int j = static_cast<int>(floorf(p.y / dx));
        int k = static_cast<int>(floorf(p.z / dx));
        if (i < 0)   i = 0;
        if (i >= W) i = W - 1;
        if (j < 0)   j = 0;
        if (j >= H) j = H - 1;
        if (k < 0)   k = 0;
        if (k >= D) k = D - 1;
        int idx = i + j * W + k * (W * H);
        if(labels[idx]!= Utility::SOLID){
            labels[idx] = Utility::FLUID;
        }
    }
};


__host__ int FluidSolver3D::labelGrid() {
    int totalCells = labels.width() * labels.height() * labels.depth();

    // 1) Для каждой ячейки: если была SOLID, остаётся SOLID; иначе – AIR
    thrust::transform(
            thrust::device,
            labels.device_data.begin(),
            labels.device_data.begin() + totalCells,
            labels.device_data.begin(),
            ClearNonSolidFunctor()
    );

    // 2) Пометка FLUID-ячееk по текущим частицам
    int numParticles = static_cast<int>(d_particles.size());
    if (numParticles > 0) {
        MarkFluidCellsFunctor functor(
                thrust::raw_pointer_cast(d_particles.data()),
                dx,
                gridWidth, gridHeight, gridDepth,
                thrust::raw_pointer_cast(labels.device_data.data())
        );
        thrust::for_each(
                thrust::device,
                thrust::make_counting_iterator<int>(0),
                thrust::make_counting_iterator<int>(numParticles),
                functor
        );
    }

    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "labelGrid3D_gpu error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }
    return 0;
}

void FluidSolver3D::saveVelocities() {
    thrust::copy(thrust::device,
                 u.device_data.begin(), u.device_data.end(),
                 uSaved.device_data.begin());

    thrust::copy(thrust::device,
                 v.device_data.begin(), v.device_data.end(),
                 vSaved.device_data.begin());

    thrust::copy(thrust::device,
                 w.device_data.begin(), w.device_data.end(),
                 wSaved.device_data.begin());
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "saveVelocities() error: " << hipGetErrorString(err) << std::endl;
    }
}

// Структура для хранения временных данных (числители и знаменатели в p2g)
struct VelocityAccumulators {
    thrust::device_vector<float> uNum, uDen;
    thrust::device_vector<float> vNum, vDen;
    thrust::device_vector<float> wNum, wDen;

    VelocityAccumulators(int uSize, int vSize, int wSize) :
            uNum(uSize), uDen(uSize),
            vNum(vSize), vDen(vSize),
            wNum(wSize), wDen(wSize)
    {
        thrust::fill(uNum.begin(), uNum.end(), 0.0f);
        thrust::fill(uDen.begin(), uDen.end(), 0.0f);
        thrust::fill(vNum.begin(), vNum.end(), 0.0f);
        thrust::fill(vDen.begin(), vDen.end(), 0.0f);
        thrust::fill(wNum.begin(), wNum.end(), 0.0f);
        thrust::fill(wDen.begin(), wDen.end(), 0.0f);
    }
};

// Функтор для накопления скоростей
struct AccumulateVelocities {
    // Размеры сетки
    int gridW, gridH, gridD;
    float dx;

    // Полушаги для MAC-узлов
    // u-грань: узлы по X имеют размер (gridW+1)×gridH×gridD
    // v-грань: узлы по Y имеют размер gridW×(gridH+1)×gridD
    // w-грань: узлы по Z имеют размер gridW×gridH×(gridD+1)

    // Сырой указатель на массив частиц
    const Utility::Particle3D* particles;
    int numParticles;

    // Указатели на временные аккумуляторы
    float* uNum;
    float* uDen;
    float* vNum;
    float* vDen;
    float* wNum;
    float* wDen;

    // Индексы stride для удобства
    // Для u: strideU_xy = (gridW+1) * gridH  (то есть шаг по Z)
    // Для v: strideV_xy = gridW * (gridH+1)
    // Для w: strideW_xy = gridW * gridH
    int strideU_z;
    int strideV_z;
    int strideW_z;

    __device__
    void operator()(int pid) const {
        // Получаем частицу
        const Utility::Particle3D& P = particles[pid];
        float3 pos = P.pos;
        float3 vel = P.vel;

        // Нормализуем к “ячейковым” координатам
        float rx = pos.x / dx;
        float ry = pos.y / dx;
        float rz = pos.z / dx;

        // --- Расчёт вкладов для компоненты U (face-centered по X) ---
        // базовые индексы iU ∈ [0..gridW], jU ∈ [0..gridH-1], kU ∈ [0..gridD-1]
        int iU = floorf(rx);
        int jU = floorf(ry - 0.5f);
        int kU = floorf(rz - 0.5f);
        iU = min(max(iU, 0),     gridW);
        jU = min(max(jU, 0),     gridH - 1);
        kU = min(max(kU, 0),     gridD - 1);

        float fxU = rx      - iU;
        float fyU = (ry - 0.5f) - jU;
        float fzU = (rz - 0.5f) - kU;
        fxU = fminf(fmaxf(fxU, 0.0f), 1.0f);
        fyU = fminf(fmaxf(fyU, 0.0f), 1.0f);
        fzU = fminf(fmaxf(fzU, 0.0f), 1.0f);

        // Проходим по 8 соседним узлам U
        for (int dz = 0; dz < 2; ++dz) {
            int k = kU + dz;
            if (k < 0 || k >= gridD) continue;
            for (int dy = 0; dy < 2; ++dy) {
                int j = jU + dy;
                if (j < 0 || j >= gridH) continue;
                for (int dxU = 0; dxU < 2; ++dxU) {
                    int i = iU + dxU;
                    if (i < 0 || i > gridW) continue;

                    float wx = (dxU == 0 ? (1.0f - fxU) : fxU);
                    float wy = (dy  == 0 ? (1.0f - fyU) : fyU);
                    float wz = (dz  == 0 ? (1.0f - fzU) : fzU);
                    float wgt = wx * wy * wz;

                    int idx = i + j * (gridW + 1) + k * strideU_z;
                    atomicAdd(&uNum[idx], vel.x * wgt);
                    atomicAdd(&uDen[idx],        wgt);
                }
            }
        }

        // --- Расчёт вкладов для компоненты V (face-centered по Y) ---
        // базовые индексы iV ∈ [0..gridW-1], jV ∈ [0..gridH], kV ∈ [0..gridD-1]
        int iV = floorf(rx - 0.5f);
        int jV = floorf(ry);
        int kV = floorf(rz - 0.5f);
        iV = min(max(iV, 0),     gridW - 1);
        jV = min(max(jV, 0),     gridH);
        kV = min(max(kV, 0),     gridD - 1);

        float fxV = (rx - 0.5f) - iV;
        float fyV = ry        - jV;
        float fzV = (rz - 0.5f) - kV;
        fxV = fminf(fmaxf(fxV, 0.0f), 1.0f);
        fyV = fminf(fmaxf(fyV, 0.0f), 1.0f);
        fzV = fminf(fmaxf(fzV, 0.0f), 1.0f);

        for (int dz = 0; dz < 2; ++dz) {
            int k = kV + dz;
            if (k < 0 || k >= gridD) continue;
            for (int dy = 0; dy < 2; ++dy) {
                int j = jV + dy;
                if (j < 0 || j > gridH) continue;
                for (int dxV = 0; dxV < 2; ++dxV) {
                    int i = iV + dxV;
                    if (i < 0 || i >= gridW) continue;

                    float wx = (dxV == 0 ? (1.0f - fxV) : fxV);
                    float wy = (dy  == 0 ? (1.0f - fyV) : fyV);
                    float wz = (dz  == 0 ? (1.0f - fzV) : fzV);
                    float wgt = wx * wy * wz;

                    int idx = i + j * gridW + k * strideV_z;
                    atomicAdd(&vNum[idx], vel.y * wgt);
                    atomicAdd(&vDen[idx],        wgt);
                }
            }
        }

        // --- Расчёт вкладов для компоненты W (face-centered по Z) ---
        // базовые индексы iW ∈ [0..gridW-1], jW ∈ [0..gridH-1], kW ∈ [0..gridD]
        int iW = floorf(rx - 0.5f);
        int jW = floorf(ry - 0.5f);
        int kW = floorf(rz);
        iW = min(max(iW, 0),     gridW - 1);
        jW = min(max(jW, 0),     gridH - 1);
        kW = min(max(kW, 0),     gridD);

        float fxW = (rx - 0.5f) - iW;
        float fyW = (ry - 0.5f) - jW;
        float fzW = rz        - kW;
        fxW = fminf(fmaxf(fxW, 0.0f), 1.0f);
        fyW = fminf(fmaxf(fyW, 0.0f), 1.0f);
        fzW = fminf(fmaxf(fzW, 0.0f), 1.0f);

        for (int dz = 0; dz < 2; ++dz) {
            int k = kW + dz;
            if (k < 0 || k > gridD) continue;
            for (int dy = 0; dy < 2; ++dy) {
                int j = jW + dy;
                if (j < 0 || j >= gridH) continue;
                for (int dxW = 0; dxW < 2; ++dxW) {
                    int i = iW + dxW;
                    if (i < 0 || i >= gridW) continue;

                    float wx = (dxW == 0 ? (1.0f - fxW) : fxW);
                    float wy = (dy  == 0 ? (1.0f - fyW) : fyW);
                    float wz = (dz  == 0 ? (1.0f - fzW) : fzW);
                    float wgt = wx * wy * wz;

                    int idx = i + j * gridW + k * strideW_z;
                    atomicAdd(&wNum[idx], vel.z * wgt);
                    atomicAdd(&wDen[idx],        wgt);
                }
            }
        }
    }
};

//функтор для вычисления скорости (делим собранные num на den)
struct ComputeVelocityFunc {
    __device__
    float operator()(const thrust::tuple<float, float>& t) const {
        float num = thrust::get<0>(t);
        float den = thrust::get<1>(t);
        return (den > 1e-8f) ? num / den : 0.0f;
    }
};

void FluidSolver3D::particlesToGrid() {
    // 3.1) Размеры компонент скорости
    const int uSize = (gridWidth + 1) * gridHeight * gridDepth;
    const int vSize = gridWidth * (gridHeight + 1) * gridDepth;
    const int wSize = gridWidth * gridHeight * (gridDepth + 1);

    // 3.2) Создаём временные аккумуляторы и обнуляем их
    VelocityAccumulators accum(uSize, vSize, wSize);

    // Заполняем нулями
    thrust::fill(accum.uNum.begin(), accum.uNum.end(), 0.0f);
    thrust::fill(accum.uDen.begin(), accum.uDen.end(), 0.0f);
    thrust::fill(accum.vNum.begin(), accum.vNum.end(), 0.0f);
    thrust::fill(accum.vDen.begin(), accum.vDen.end(), 0.0f);
    thrust::fill(accum.wNum.begin(), accum.wNum.end(), 0.0f);
    thrust::fill(accum.wDen.begin(), accum.wDen.end(), 0.0f);

    // 3.3) Настраиваем функтор накопления
    AccumulateVelocities accFunc;
    accFunc.dx        = dx;
    accFunc.gridW     = gridWidth;
    accFunc.gridH     = gridHeight;
    accFunc.gridD     = gridDepth;
    accFunc.particles = thrust::raw_pointer_cast(d_particles.data());
    accFunc.numParticles = static_cast<int>(d_particles.size());
    accFunc.uNum = thrust::raw_pointer_cast(accum.uNum.data());
    accFunc.uDen = thrust::raw_pointer_cast(accum.uDen.data());
    accFunc.vNum = thrust::raw_pointer_cast(accum.vNum.data());
    accFunc.vDen = thrust::raw_pointer_cast(accum.vDen.data());
    accFunc.wNum = thrust::raw_pointer_cast(accum.wNum.data());
    accFunc.wDen = thrust::raw_pointer_cast(accum.wDen.data());

    // Вычисляем strides (шаг по Z) для каждого массива
    accFunc.strideU_z = (gridWidth + 1) * gridHeight;
    accFunc.strideV_z =  gridWidth       * (gridHeight + 1);
    accFunc.strideW_z =  gridWidth       *  gridHeight;

    // 3.4) Запускаем накопление атомарными операциями
    thrust::for_each(
            thrust::device,
            thrust::counting_iterator<int>(0),
            thrust::counting_iterator<int>(static_cast<int>(d_particles.size())),
            accFunc
    );

    // 3.5) Нормализуем (num/den) → записываем в u,v,w
    // Для u-компоненты
    thrust::transform(
            thrust::device,
            thrust::make_zip_iterator(thrust::make_tuple(accum.uNum.begin(), accum.uDen.begin())),
            thrust::make_zip_iterator(thrust::make_tuple(accum.uNum.end(),   accum.uDen.end())),
            u.device_data.begin(),
            ComputeVelocityFunc()
    );

    // Для v-компоненты
    thrust::transform(
            thrust::device,
            thrust::make_zip_iterator(thrust::make_tuple(accum.vNum.begin(), accum.vDen.begin())),
            thrust::make_zip_iterator(thrust::make_tuple(accum.vNum.end(),   accum.vDen.end())),
            v.device_data.begin(),
            ComputeVelocityFunc()
    );

    // Для w-компоненты
    thrust::transform(
            thrust::device,
            thrust::make_zip_iterator(thrust::make_tuple(accum.wNum.begin(), accum.wDen.begin())),
            thrust::make_zip_iterator(thrust::make_tuple(accum.wNum.end(),   accum.wDen.end())),
            w.device_data.begin(),
            ComputeVelocityFunc()
    );

    // 3.6) Проверяем ошибки CUDA
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "P2G error: " << hipGetErrorString(err) << std::endl;
    }
}

struct ApplyScalarForce
{
    float dt, a, vel_unknown;

    ApplyScalarForce(float _dt, float _a, float _vel_unknown)
            : dt(_dt), a(_a), vel_unknown(_vel_unknown) {}

    __host__ __device__
    float operator()(const float& x) const {
        return (x > vel_unknown) ? x + dt * a : x;
    }
};

void FluidSolver3D::applyForces(){

    thrust::transform(
            thrust::device,
            u.device_data.begin(), u.device_data.end(),         // вход
            u.device_data.begin(),                  // выход
            ApplyScalarForce(dt, GRAVITY.x, VEL_UNKNOWN)
    );
//    u.copy_to_host();
//        std::cout << "----u 3d---" << std::endl;
//    for(int k = 0; k < gridDepth; ++k){
//        for(int j = 0; j < gridHeight; ++j){
//            for(int i = 0; i < gridWidth+1; ++i){
//                std::cout << u.host_data[i + j*(gridWidth+1) + k * (gridWidth+1)*gridHeight] << ", ";
//            }
//            std::cout << std::endl;
//        }
//        std::cout << std::endl;
//    }

    thrust::transform(
            thrust::device,
            v.device_data.begin(), v.device_data.end(),
            v.device_data.begin(),
            ApplyScalarForce(dt, GRAVITY.y, VEL_UNKNOWN)
    );

//    v.copy_to_host();
//    std::cout << "----v 3d---" << std::endl;
//    for(int k = 0; k < gridDepth; ++k){
//        for(int j = 0; j < gridHeight+1; ++j){
//            for(int i = 0; i < gridWidth; ++i){
//                std::cout << v.host_data[i + j*(gridWidth) + k * (gridWidth)*(gridHeight+1)] << ", ";
//            }
//            std::cout << std::endl;
//        }
//        std::cout << std::endl;
//    }

    thrust::transform(
            thrust::device,
            w.device_data.begin(), w.device_data.end(),
            w.device_data.begin(),
            ApplyScalarForce(dt, GRAVITY.z, VEL_UNKNOWN)
    );

//    w.copy_to_host();
//    std::cout << "----w 3d---" << std::endl;
//    for(int k = 0; k < gridDepth+1; ++k){
//        for(int j = 0; j < gridHeight; ++j){
//            for(int i = 0; i < gridWidth; ++i){
//                std::cout << w.host_data[i + j*(gridWidth) + k * (gridWidth)*(gridHeight)] << ", ";
//            }
//            std::cout << std::endl;
//        }
//        std::cout << std::endl;
//    }
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "ApplyForces() error: " << hipGetErrorString(err) << std::endl;
    }
}

// ----------------------------------
// 1) Вспомогательная __device__-функция трёхлинейной интерполяции
__host__ __device__
float trilerp(const float* fld,
              int i,int j,int k,
              float fx,float fy,float fz,
              int stride_i,int stride_j)
{
#define SAMP(ii,jj,kk) fld[(ii) + (jj)*stride_i + (kk)*stride_j]
    float c00 = SAMP(i  , j  , k  )*(1-fx) + SAMP(i+1, j  , k  )*fx;
    float c10 = SAMP(i  , j+1, k  )*(1-fx) + SAMP(i+1, j+1, k  )*fx;
    float c01 = SAMP(i  , j  , k+1)*(1-fx) + SAMP(i+1, j  , k+1)*fx;
    float c11 = SAMP(i  , j+1, k+1)*(1-fx) + SAMP(i+1, j+1, k+1)*fx;
    float c0  = c00*(1-fy) + c10*fy;
    float c1  = c01*(1-fy) + c11*fy;
    float c   = c0*(1-fz) + c1*fz;
#undef SAMP
    return c;
}

__host__ __device__
float trilinearInterpolation(
        const float xd, const float yd, const float zd,
        const float c000, const float c100, const float c001, const float c101,
        const float c010, const float c110, const float c011, const float c111)
{
    // Интерполяция вдоль X
    float c00 = c000 * (1.0f - xd) + c100 * xd;  // z=0, y=0
    float c01 = c001 * (1.0f - xd) + c101 * xd;  // z=1, y=0
    float c10 = c010 * (1.0f - xd) + c110 * xd;  // z=0, y=1
    float c11 = c011 * (1.0f - xd) + c111 * xd;  // z=1, y=1

    // Интерполяция вдоль Y
    float c0 = c00 * (1.0f - yd) + c10 * yd;  // z=0
    float c1 = c01 * (1.0f - yd) + c11 * yd;  // z=1

    // Интерполяция вдоль Z
    return c0 * (1.0f - zd) + c1 * zd;
}
// ----------------------------------
struct GridToParticleFunctor
{
    int W, H, D;
    float  dx, alpha;

    // raw-указатели на device_data[]
    const float *u, *v, *w;
    const float *du, *dv, *dw;

    GridToParticleFunctor(int _W, int _H, int _D,
                          float _dx, float _alpha,
                          const float* _u, const float* _v, const float* _w,
                          const float* _du, const float* _dv, const float* _dw)
            : W(_W), H(_H), D(_D),
              dx(_dx), alpha(_alpha),
              u(_u), v(_v), w(_w),
              du(_du), dv(_dv), dw(_dw) {}

    __device__
    Utility::Particle3D operator()(const Utility::Particle3D& pin) const
    {
        Utility::Particle3D pout = pin;

        // 1) Нормализованные “cell-space” координаты
        float rx = pin.pos.x / dx;
        float ry = pin.pos.y / dx;
        float rz = pin.pos.z / dx;

        // --- Интерполяция U (MAC face по X) ---
        // базовые индексы: iU ∈ [0..W], jU ∈ [0..H-1], kU ∈ [0..D-1]
        int iU = floorf(rx);
        int jU = floorf(ry - 0.5f);
        int kU = floorf(rz - 0.5f);
        // зажимаем в допустимый диапазон
        iU = min(max(iU, 0),     W);
        jU = min(max(jU, 0),     H - 1);
        kU = min(max(kU, 0),     D - 1);
        // дробные части внутри “ячейки” U (координаты относительно ячейки)
        float fxU = rx - iU;
        float fyU = (ry - 0.5f) - jU;
        float fzU = (rz - 0.5f) - kU;
        fxU = fminf(fmaxf(fxU, 0.0f), 1.0f);
        fyU = fminf(fmaxf(fyU, 0.0f), 1.0f);
        fzU = fminf(fmaxf(fzU, 0.0f), 1.0f);

        // адресация 8 вершин массива u (размер (W+1) × H × D):
        // idx_u(i,j,k) = i + j*(W+1) + k*(W+1)*H
        int baseU = jU * (W + 1) + kU * (W + 1) * H;
        float u000 = u[ iU     + baseU ];
        float u100 = u[(iU + 1) + baseU ];
        float u010 = u[ iU     + (jU + 1) * (W + 1) + kU * (W + 1) * H ];
        float u110 = u[(iU + 1) + (jU + 1) * (W + 1) + kU * (W + 1) * H ];
        float u001 = u[ iU     + jU * (W + 1) + (kU + 1) * (W + 1) * H ];
        float u101 = u[(iU + 1) + jU * (W + 1) + (kU + 1) * (W + 1) * H ];
        float u011 = u[ iU     + (jU + 1) * (W + 1) + (kU + 1) * (W + 1) * H ];
        float u111 = u[(iU + 1) + (jU + 1) * (W + 1) + (kU + 1) * (W + 1) * H ];
        float uPIC = trilinearInterpolation(
                fxU, fyU, fzU,
                u000, u100, u001, u101,
                u010, u110, u011, u111
        );

        float du000 = du[ iU     + baseU ];
        float du100 = du[(iU + 1) + baseU ];
        float du010 = du[ iU     + (jU + 1) * (W + 1) + kU * (W + 1) * H ];
        float du110 = du[(iU + 1) + (jU + 1) * (W + 1) + kU * (W + 1) * H ];
        float du001 = du[ iU     + jU * (W + 1) + (kU + 1) * (W + 1) * H ];
        float du101 = du[(iU + 1) + jU * (W + 1) + (kU + 1) * (W + 1) * H ];
        float du011 = du[ iU     + (jU + 1) * (W + 1) + (kU + 1) * (W + 1) * H ];
        float du111 = du[(iU + 1) + (jU + 1) * (W + 1) + (kU + 1) * (W + 1) * H ];
        float duFLIP = trilinearInterpolation(
                fxU, fyU, fzU,
                du000, du100, du001, du101,
                du010, du110, du011, du111
        );

        // --- Интерполяция V (MAC face по Y) ---
        // iV ∈ [0..W-1], jV ∈ [0..H], kV ∈ [0..D-1]
        int iV = floorf(rx - 0.5f);
        int jV = floorf(ry);
        int kV = floorf(rz - 0.5f);
        iV = min(max(iV, 0),     W - 1);
        jV = min(max(jV, 0),     H);
        kV = min(max(kV, 0),     D - 1);

        float fxV = (rx - 0.5f) - iV;
        float fyV = ry - jV;
        float fzV = (rz - 0.5f) - kV;
        fxV = fminf(fmaxf(fxV, 0.0f), 1.0f);
        fyV = fminf(fmaxf(fyV, 0.0f), 1.0f);
        fzV = fminf(fmaxf(fzV, 0.0f), 1.0f);

        // idx_v(i,j,k) = i + j*W + k*(W*(H+1))
        int baseV = jV * W + kV * (W * (H + 1));
        float v000 = v[ iV     + baseV ];
        float v100 = v[(iV + 1) + baseV ];
        float v010 = v[ iV     + (jV + 1) * W + kV * (W * (H + 1)) ];
        float v110 = v[(iV + 1) + (jV + 1) * W + kV * (W * (H + 1)) ];
        float v001 = v[ iV     + jV * W + (kV + 1) * (W * (H + 1)) ];
        float v101 = v[(iV + 1) + jV * W + (kV + 1) * (W * (H + 1)) ];
        float v011 = v[ iV     + (jV + 1) * W + (kV + 1) * (W * (H + 1)) ];
        float v111 = v[(iV + 1) + (jV + 1) * W + (kV + 1) * (W * (H + 1)) ];
        float vPIC = trilinearInterpolation(
                fxV, fyV, fzV,
                v000, v100, v001, v101,
                v010, v110, v011, v111
        );

        float dv000 = dv[ iV     + baseV ];
        float dv100 = dv[(iV + 1) + baseV ];
        float dv010 = dv[ iV     + (jV + 1) * W + kV * (W * (H + 1)) ];
        float dv110 = dv[(iV + 1) + (jV + 1) * W + kV * (W * (H + 1)) ];
        float dv001 = dv[ iV     + jV * W + (kV + 1) * (W * (H + 1)) ];
        float dv101 = dv[(iV + 1) + jV * W + (kV + 1) * (W * (H + 1)) ];
        float dv011 = dv[ iV     + (jV + 1) * W + (kV + 1) * (W * (H + 1)) ];
        float dv111 = dv[(iV + 1) + (jV + 1) * W + (kV + 1) * (W * (H + 1)) ];
        float dvFLIP = trilinearInterpolation(
                fxV, fyV, fzV,
                dv000, dv100, dv001, dv101,
                dv010, dv110, dv011, dv111
        );

        // --- Интерполяция W (MAC face по Z) ---
        // iW ∈ [0..W-1], jW ∈ [0..H-1], kW ∈ [0..D]
        int iW = floorf(rx - 0.5f);
        int jW = floorf(ry - 0.5f);
        int kW = floorf(rz);
        iW = min(max(iW, 0),     W - 1);
        jW = min(max(jW, 0),     H - 1);
        kW = min(max(kW, 0),     D);

        float fxW = (rx - 0.5f) - iW;
        float fyW = (ry - 0.5f) - jW;
        float fzW = rz - kW;
        fxW = fminf(fmaxf(fxW, 0.0f), 1.0f);
        fyW = fminf(fmaxf(fyW, 0.0f), 1.0f);
        fzW = fminf(fmaxf(fzW, 0.0f), 1.0f);

        // idx_w(i,j,k) = i + j*W + k*(W*H)
        int baseW = jW * W + kW * (W * H);
        float w000 = w[ iW     + baseW ];
        float w100 = w[(iW + 1) + baseW ];
        float w010 = w[ iW     + (jW + 1) * W + kW * (W * H) ];
        float w110 = w[(iW + 1) + (jW + 1) * W + kW * (W * H) ];
        float w001 = w[ iW     + jW * W + (kW + 1) * (W * H) ];
        float w101 = w[(iW + 1) + jW * W + (kW + 1) * (W * H) ];
        float w011 = w[ iW     + (jW + 1) * W + (kW + 1) * (W * H) ];
        float w111 = w[(iW + 1) + (jW + 1) * W + (kW + 1) * (W * H) ];
        float wPIC = trilinearInterpolation(
                fxW, fyW, fzW,
                w000, w100, w001, w101,
                w010, w110, w011, w111
        );

        float dw000 = dw[ iW     + baseW ];
        float dw100 = dw[(iW + 1) + baseW ];
        float dw010 = dw[ iW     + (jW + 1) * W + kW * (W * H) ];
        float dw110 = dw[(iW + 1) + (jW + 1) * W + kW * (W * H) ];
        float dw001 = dw[ iW     + jW * W + (kW + 1) * (W * H) ];
        float dw101 = dw[(iW + 1) + jW * W + (kW + 1) * (W * H) ];
        float dw011 = dw[ iW     + (jW + 1) * W + (kW + 1) * (W * H) ];
        float dw111 = dw[(iW + 1) + (jW + 1) * W + (kW + 1) * (W * H) ];
        float dwFLIP = trilinearInterpolation(
                fxW, fyW, fzW,
                dw000, dw100, dw001, dw101,
                dw010, dw110, dw011, dw111
        );

        // 2) Форма FLIP: pin.vel + delta-скорость, а PIC – это просто uPIC,vPIC,wPIC
        float newU = alpha * uPIC + (1.0f - alpha) * (pin.vel.x + duFLIP);
        float newV = alpha * vPIC + (1.0f - alpha) * (pin.vel.y + dvFLIP);
        float newW = alpha * wPIC + (1.0f - alpha) * (pin.vel.z + dwFLIP);

        pout.vel = make_float3(newU, newV, newW);

        return pout;
    }
};

// ----------------------------------
void FluidSolver3D::gridToParticles(float alpha)
{
    // 1) размеры MAC-решёток
    int Nu = (gridWidth + 1) * gridHeight * gridDepth;
    int Nv = gridWidth * (gridHeight + 1) * gridDepth;
    int Nw = gridWidth * gridHeight * (gridDepth + 1);

    // 2) вычисляем дельты (new – old) в отдельные device_vector
    thrust::device_vector<float> du(Nu);
    thrust::device_vector<float> dv(Nv);
    thrust::device_vector<float> dw(Nw);

    thrust::transform(
            u.device_data.begin(), u.device_data.end(),
            uSaved.device_data.begin(),
            du.begin(),
            thrust::minus<float>()
    );
    thrust::transform(
            v.device_data.begin(), v.device_data.end(),
            vSaved.device_data.begin(),
            dv.begin(),
            thrust::minus<float>()
    );
    thrust::transform(
            w.device_data.begin(), w.device_data.end(),
            wSaved.device_data.begin(),
            dw.begin(),
            thrust::minus<float>()
    );

    // 3) получаем raw-указатели
    const float* pu  = u.device_ptr();
    const float* pv  = v.device_ptr();
    const float* pw  = w.device_ptr();
    const float* pdu = thrust::raw_pointer_cast(du.data());
    const float* pdv = thrust::raw_pointer_cast(dv.data());
    const float* pdw = thrust::raw_pointer_cast(dw.data());

    // 4) запускаем один transform по всем частицам
    thrust::transform(
            d_particles.begin(),
            d_particles.end(),
            d_particles.begin(),
            GridToParticleFunctor(
                    gridWidth, gridHeight, gridDepth,
                    dx, alpha,
                    pu, pv, pw,
                    pdu, pdv, pdw
            )
    );

    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "G2P error: " << hipGetErrorString(err) << std::endl;
    }
}

__device__ inline bool isCellValid(int x, int y, int z, int W, int H, int D) {
    return x >= 0 && x < W && y >= 0 && y < H && z >= 0 && z < D;
}

__device__ inline int idx3d(int x , int y, int z, int W, int H){
    return x + y * W + z * W * H;
}

// ----------------------------------
// 1) Исправленная функция интерполяции MAC-скоростей (без FLIP-дельт)
__device__
float3 interpVelDevice3D(const float* u, const float* v, const float* w,
                         int W, int H, int D, float dx, float3 pos)
{
    // Преобразуем позицию в “ячейковые” координаты
    float rx = pos.x / dx;
    float ry = pos.y / dx;
    float rz = pos.z / dx;

    // --- Интерполяция U (MAC-грань по X) ---
    // iU ∈ [0..W], jU ∈ [0..H-1], kU ∈ [0..D-1]
    int iU = floorf(rx);
    int jU = floorf(ry - 0.5f);
    int kU = floorf(rz - 0.5f);
    iU = min(max(iU, 0),     W);
    jU = min(max(jU, 0),     H - 1);
    kU = min(max(kU, 0),     D - 1);

    float fxU = rx - iU;
    float fyU = (ry - 0.5f) - jU;
    float fzU = (rz - 0.5f) - kU;
    fxU = fminf(fmaxf(fxU, 0.0f), 1.0f);
    fyU = fminf(fmaxf(fyU, 0.0f), 1.0f);
    fzU = fminf(fmaxf(fzU, 0.0f), 1.0f);

    // Линейный индекс в массиве u: idx_u(i,j,k) = i + j*(W+1) + k*(W+1)*H
    int baseU = jU * (W + 1) + kU * (W + 1) * H;
    float u000 = u[ iU     + baseU ];
    float u100 = u[(iU + 1) + baseU ];
    float u010 = u[ iU     + (jU + 1)*(W + 1) + kU*(W + 1)*H ];
    float u110 = u[(iU + 1) + (jU + 1)*(W + 1) + kU*(W + 1)*H ];
    float u001 = u[ iU     + jU*(W + 1) + (kU + 1)*(W + 1)*H ];
    float u101 = u[(iU + 1) + jU*(W + 1) + (kU + 1)*(W + 1)*H ];
    float u011 = u[ iU     + (jU + 1)*(W + 1) + (kU + 1)*(W + 1)*H ];
    float u111 = u[(iU + 1) + (jU + 1)*(W + 1) + (kU + 1)*(W + 1)*H ];
    float uInterp = trilinearInterpolation(
            fxU, fyU, fzU,
            u000, u100, u001, u101,
            u010, u110, u011, u111
    );

    // --- Интерполяция V (MAC-грань по Y) ---
    // iV ∈ [0..W-1], jV ∈ [0..H], kV ∈ [0..D-1]
    int iV = floorf(rx - 0.5f);
    int jV = floorf(ry);
    int kV = floorf(rz - 0.5f);
    iV = min(max(iV, 0),     W - 1);
    jV = min(max(jV, 0),     H);
    kV = min(max(kV, 0),     D - 1);

    float fxV = (rx - 0.5f) - iV;
    float fyV = ry - jV;
    float fzV = (rz - 0.5f) - kV;
    fxV = fminf(fmaxf(fxV, 0.0f), 1.0f);
    fyV = fminf(fmaxf(fyV, 0.0f), 1.0f);
    fzV = fminf(fmaxf(fzV, 0.0f), 1.0f);

    // idx_v(i,j,k) = i + j*W + k*(W*(H+1))
    int baseV = jV * W + kV * (W * (H + 1));
    float v000 = v[ iV     + baseV ];
    float v100 = v[(iV + 1) + baseV ];
    float v010 = v[ iV     + (jV + 1)*W + kV*(W*(H + 1)) ];
    float v110 = v[(iV + 1) + (jV + 1)*W + kV*(W*(H + 1)) ];
    float v001 = v[ iV     + jV*W + (kV + 1)*(W*(H + 1)) ];
    float v101 = v[(iV + 1) + jV*W + (kV + 1)*(W*(H + 1)) ];
    float v011 = v[ iV     + (jV + 1)*W + (kV + 1)*(W*(H + 1)) ];
    float v111 = v[(iV + 1) + (jV + 1)*W + (kV + 1)*(W*(H + 1)) ];
    float vInterp = trilinearInterpolation(
            fxV, fyV, fzV,
            v000, v100, v001, v101,
            v010, v110, v011, v111
    );

    // --- Интерполяция W (MAC-грань по Z) ---
    // iW ∈ [0..W-1], jW ∈ [0..H-1], kW ∈ [0..D]
    int iW = floorf(rx - 0.5f);
    int jW = floorf(ry - 0.5f);
    int kW = floorf(rz);
    iW = min(max(iW, 0),     W - 1);
    jW = min(max(jW, 0),     H - 1);
    kW = min(max(kW, 0),     D);

    float fxW = (rx - 0.5f) - iW;
    float fyW = (ry - 0.5f) - jW;
    float fzW = rz - kW;
    fxW = fminf(fmaxf(fxW, 0.0f), 1.0f);
    fyW = fminf(fmaxf(fyW, 0.0f), 1.0f);
    fzW = fminf(fmaxf(fzW, 0.0f), 1.0f);

    // idx_w(i,j,k) = i + j*W + k*(W*H)
    int baseW = jW * W + kW * (W * H);
    float w000 = w[ iW     + baseW ];
    float w100 = w[(iW + 1) + baseW ];
    float w010 = w[ iW     + (jW + 1)*W + kW*(W*H) ];
    float w110 = w[(iW + 1) + (jW + 1)*W + kW*(W*H) ];
    float w001 = w[ iW     + jW*W + (kW + 1)*(W*H) ];
    float w101 = w[(iW + 1) + jW*W + (kW + 1)*(W*H) ];
    float w011 = w[ iW     + (jW + 1)*W + (kW + 1)*(W*H) ];
    float w111 = w[(iW + 1) + (jW + 1)*W + (kW + 1)*(W*H) ];
    float wInterp = trilinearInterpolation(
            fxW, fyW, fzW,
            w000, w100, w001, w101,
            w010, w110, w011, w111
    );

    return make_float3(uInterp, vInterp, wInterp);
}

// ----------------------------------
// 2) Исправленный функтор явной адвекции (Runge-Kutta / Heun не нужен — используем адаптивный Эйлер)
__device__
bool projectParticleDevice3D(Utility::Particle3D &particle,
                             const int* labels,
                             int W, int H, int D, float dx)
{
    // 26 соседей
    const int off[26][3] = {
            { 1, 0, 0}, {-1, 0, 0},
            { 0, 1, 0}, { 0,-1, 0},
            { 0, 0, 1}, { 0, 0,-1},
            {1, 1, 0}, {-1, 1, 0},
            {1, -1, 0}, {-1, -1},
            {1, 0, 1}, {-1, 0, 1},
            {1, 0, -1}, {-1, 0, -1},
            {0, 1, 1}, {0, -1, 1},
            {0, 1, -1}, {0, -1, -1},
            {1, 1, 1}, {-1, 1, 1},
            {1, -1, 1}, {1, 1, -1},
            {-1, -1, 1}, {-1, 1, -1},
            {1, -1, -1}, {-1, -1, -1}
    };

    // Текущая клетка
    int cx = int(floorf(particle.pos.x / dx));
    int cy = int(floorf(particle.pos.y / dx));
    int cz = int(floorf(particle.pos.z / dx));

    float3 bestPos = particle.pos;
    float  bestD   = fmaxf(W, H) * dx; //1e10f;
    bool   found   = false;
    int foundNeigInd = 0;
    //ищем наименьшее расстояние до твёрдой ячейки, не нашли, тогда до ближайшей воздушной
    // Сначала пытаемся найти соседнюю клетку со статусом FLUID, потом AIR
    for (int pass = 0; pass < 2; ++pass) {
        int wanted = (pass == 0 ? Utility::FLUID : Utility::AIR);
        for (int n = 0; n < 26; ++n) {
            int nx = cx + off[n][0];
            int ny = cy + off[n][1];
            int nz = cz + off[n][2];
            if (nx < 0 || nx >= W || ny < 0 || ny >= H || nz < 0 || nz >= D) continue; 
            int idx = nx + ny*W + nz*W*H;
            if (labels[idx] != wanted) continue;

            float3 cellC = make_float3(
                    (nx + 0.5f) * dx,
                    (ny + 0.5f) * dx,
                    (nz + 0.5f) * dx
            );  //координаты центра ячейки
            float d = (cellC.x - particle.pos.x) * (cellC.x - particle.pos.x)
                      + (cellC.y - particle.pos.y) * (cellC.y - particle.pos.y)
                      + (cellC.z - particle.pos.z) * (cellC.z - particle.pos.z); //квадрат расстояния от положения частицы до центра ячейки 
            if (d < bestD) {
                bestD   = d;
                bestPos = cellC;
                found   = true;
                foundNeigInd = n;
            }
        }
        if (found) break;
    }
    if (!found) return false;

    // Переносим частицу на центр найденной соседней клетки
    particle.pos = bestPos;

    // во избежание накопления частиц в центрах ячеек:
    thrust::default_random_engine randEng;
    thrust::uniform_real_distribution<float> uniDist(-1.0f, 1.0f);
    randEng.discard(foundNeigInd);
    particle.pos.x += uniDist(randEng) * 0.25f * dx;
    particle.pos.y += uniDist(randEng) * 0.25f * dx;
    particle.pos.z += uniDist(randEng) * 0.25f * dx;
    return true;
}

struct AdvectParticlesFunctor {
    float dt, dx, C;
    int W, H, D;
    const float* u;
    const float* v;
    const float* w;
    const int*   labels;

    __host__ __device__
    AdvectParticlesFunctor(float _dt, float _dx, float _C,
                           int _W, int _H, int _D,
                           const float* _u, const float* _v, const float* _w,
                           const int*   _labels)
            : dt(_dt), dx(_dx), C(_C),
              W(_W), H(_H), D(_D),
              u(_u), v(_v), w(_w),
              labels(_labels) {}

    __device__
    Utility::Particle3D operator()(const Utility::Particle3D& pin) const {
        Utility::Particle3D particle = pin;
        float subT = 0.0f; //локальный отсчёт времени (глобальный шаг dt дробим на шаги dT)
        bool  finished = false;
        int   iter = 0;

        while (!finished && iter++ < 100) {
            // 1) Интерполируем скорость из MAC-поля
            float3 vel = interpVelDevice3D(u, v, w, W, H, D, dx, particle.pos);
                
            // 2) Рассчитываем шаг dT по CFL-критерию
            //" It has been suggested[FF01] that an appropriate strategy is to limit dT so that the furthest a particle trajectory is traced is five grid cell widths:
            float speed = sqrtf(vel.x*vel.x + vel.y*vel.y + vel.z*vel.z) +1e-37f/*+ sqrtf(5.0f * dx * 9.8f)*/; 
            float dT = (C * dx) / speed; //шаг по времени находим из критерия Куранта
            if (subT + dT >= dt) {
                dT = dt - subT;
                finished = true;
            } else if (subT + 2*dT >= dt) {
                dT *= 0.5f; // делим пополам, чтобы не выйти за dt
            }

            // 3) Явный Эйлер (возможно, стоит поменять на RK3, как это советует R. Bridson...)
            particle.pos.x += vel.x * dT;
            particle.pos.y += vel.y * dT;
            particle.pos.z += vel.z * dT;
            subT += dT;

            particle.pos.x = fmaxf(particle.pos.x, 0.0f);
            particle.pos.x = fminf(particle.pos.x, (W-1)*dx);
            particle.pos.y = fmaxf(particle.pos.y, 0.0f);
            particle.pos.y = fminf(particle.pos.y, (H-1)*dx);
            particle.pos.z = fmaxf(particle.pos.z, 0.0f);
            particle.pos.z = fminf(particle.pos.z, (D-1)*dx);

            // 4) Проверяем выход за нижние границы и NaN
            if (particle.pos.x < 0.0f || particle.pos.y < 0.0f || particle.pos.z < 0.0f ||
                isnan(particle.pos.x)  || isnan(particle.pos.y)  || isnan(particle.pos.z)) {
                    //возможно, стоит придумать, как обрабатывать такой случай
                break;
            }

            // 5) Если частица попала в SOLID-клетку, пытаемся спроецировать её в соседнюю
            int cx = int(floorf(particle.pos.x / dx));
            int cy = int(floorf(particle.pos.y / dx));
            int cz = int(floorf(particle.pos.z / dx));
            if (cx >= 0 && cx < W && cy >= 0 && cy < H && cz >= 0 && cz < D) {
                int idx = cx + cy*W + cz*W*H;
                if (labels[idx] == Utility::SOLID) {//проверка на попадание в твёрдую границу 
                    if (!projectParticleDevice3D(particle, labels, W, H, D, dx)) //насильно отбрасываем в FLUID (приоритетнее) или в AIR ячейку
                        break;
                }
            }
        }

        return particle;
    }
};

// ----------------------------------
// 3) Обновлённый метод FluidSolver3D::advectParticles
void FluidSolver3D::advectParticles(float C)
{
    const float* pu = u.device_ptr();
    const float* pv = v.device_ptr();
    const float* pw = w.device_ptr();
    const int*   pl = labels.device_ptr();

    thrust::transform(
            d_particles.begin(),
            d_particles.end(),
            d_particles.begin(),
            AdvectParticlesFunctor(
                    dt, dx, C,
                    gridWidth, gridHeight, gridDepth,
                    pu, pv, pw,
                    pl
            )
    );

    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "advectParticles() error: " << hipGetErrorString(err) << std::endl;
    }
}


struct RHSCalculator3D {
    const int* labels;
    const float* u, *v, *w;
    float scale;
    int W, H, D;
    float* rhs_temp;

    __device__ void operator()(int idx) const {
        int i = idx % W;
        int j = (idx / W) % H;
        int k = idx / (W * H);

        if (labels[idx] != Utility::FLUID) return;

        float div =
                u[(i+1) + j*(W+1) + k*(W+1)*H] - u[i + j*(W+1) + k*(W+1)*H] +
                v[i + (j+1)*W + k*W*(H+1)] - v[i + j*W + k*W*(H+1)] +
                w[i + j*W + (k+1)*W*H] - w[i + j*W + k*W*H];

        float rhs_val = -scale * div;

        // Solid boundaries
        if (i-1 >= 0 && labels[idx-1] == Utility::SOLID)
            rhs_val -= scale * (u[i + j*(W+1) + k*(W+1)*H] - 0.0f); //change 0.0f to solid boundary vel 
        if (i < W && labels[idx+1] == Utility::SOLID)
            rhs_val += scale * (u[(i+1) + j*(W+1) + k*(W+1)*H] - 0.0f);
        if (j-1 >= 0 && labels[idx-W] == Utility::SOLID)
            rhs_val -= scale * (v[i + j*W + k*W*(H+1)] - 0.0f);
        if (j < H && labels[idx+W] == Utility::SOLID)
            rhs_val += scale * (v[i + (j+1)*W + k*W*(H+1)] - 0.0f);
        if (k-1 >= 0 && labels[idx-W*H] == Utility::SOLID)
            rhs_val -= scale * (w[i + j*W + k*W*H] - 0.0f);
        if (k < D && labels[idx+W*H] == Utility::SOLID)
            rhs_val += scale * (w[i + j*W + (k+1)*W*H] - 0.0f);

        rhs_temp[idx] = rhs_val;

    }
};

// Копируем только FLUID ячейки
struct CopyFluidRHSFunctor {
    const int* fluidNumbers;
    const float* rhs_temp;

    __device__ float operator()(int idx) const {
        int fnum = fluidNumbers[idx];
        return (fnum >= 0) ? rhs_temp[idx] : 0.0f;
    }
};


struct FluidCellPredicate {
    const int* labels;
    const int FLUID;

    __device__
    bool operator()(int idx) const {
        return labels[idx] == FLUID;
    }
};


struct IsSelected {
    const int* flags; // Указатель на данные вектора меток

    IsSelected(const int* flags_ptr) : flags(flags_ptr) {}

    __host__ __device__
    bool operator()(const int idx) const {
        return flags[idx] == 1; // Возвращает true, если метка равна 1
    }
};

// Функтор-трансформатор (просто возвращает значение)
struct ValueTransformer {
    __host__ __device__
    float operator()(float val) const {
        return val;
    }
};

void FluidSolver3D::constructRHS(thrust::device_vector<float>& rhs, const thrust::device_vector<int>& fluidNumbers, const thrust::device_vector<int>& fluidFlags) {

    float scale = (FLUID_DENSITY * dx) / dt;
    //std::cout << "scale = " << scale << std::endl;
    thrust::device_vector<float> rhs_temp(w_x_h_x_d, 0.0f);

    thrust::for_each_n(
            thrust::device,
            thrust::counting_iterator<int>(0),
            w_x_h_x_d,
            RHSCalculator3D{
                    labels.device_ptr(),
                    u.device_ptr(),
                    v.device_ptr(),
                    w.device_ptr(),
                    scale,
                    gridWidth, gridHeight, gridDepth,
                    thrust::raw_pointer_cast(rhs_temp.data())
            }
    );

//    std::cout << "rhs_temp:"<<std::endl;
//    thrust::host_vector<float> rhs_temp_h = rhs_temp;
//    for(int j = 0; j < gridHeight; ++j){
//        for(int i  =0 ; i < gridWidth ; ++i){
//            std::cout << rhs_temp_h[i + j * gridWidth] << ", ";
//        }
//        std::cout << std::endl;
//    }


    const int result_size = thrust::count(fluidFlags.begin(), fluidFlags.end(), 1);
    rhs.resize(result_size);

    const int* flags_ptr = thrust::raw_pointer_cast(fluidFlags.data());
    thrust::copy_if(
            thrust::device,
            rhs_temp.begin(),
            rhs_temp.end(),
            thrust::counting_iterator<size_t>(0),
            rhs.begin(),
            IsSelected( flags_ptr)
    );

     //Вывод результата
//    thrust::host_vector<float> rhs_h = rhs;
//    //rhs = thrust::device_vector<float>{4905, 4905, 4905, -4905, -4905, -4905};
//    std::cout << "Copied values: ";
//    for (float val : rhs_h) {
//        std::cout << val << " ";
//    }
//    std::cout << std::endl;
//    std::cout << "----" << std::endl;
//    thrust::transform(
//            thrust::device,
//            thrust::counting_iterator<int>(0),
//            thrust::counting_iterator<int>(w_x_h_x_d),
//            rhs.begin(),
//            CopyFluidRHSFunctor{
//                    thrust::raw_pointer_cast(fluidNumbers.data()),
//                    thrust::raw_pointer_cast(rhs_temp.data())
//            }
//    );

    /* ДО (пример)
        Индексы:    [0]     [1]     [2]     [3]
        rhs_temp:  [1.0]  [2.0]  [3.0]  [4.0]
        labels:    [SOLID] [FLUID] [AIR] [FLUID]
     * */
    /* После (пример)
        rhs: [2.0] [4.0]  // Только FLUID-ячейки
        fluidCellsAmount = 2
     * */

}

struct MatrixBuilder3D {
    int W, H, D;
    const int* labels;        // метки ячеек
    const int* fluidNumbers;   // mapping global idx → local idx (или −1)
    int*       nnz_per_row;    // выход: сколько ненулей у строки “row”

    __device__ void operator()(int idx) const {
        // 1) перевод idx → (i,j,k)
        int i = idx % W;
        int j = (idx / W) % H;
        int k = idx / (W * H);

        // 2) работаем только для FLUID-ячейки
        if (labels[idx] != Utility::FLUID) return;

        int row = fluidNumbers[idx];   // локальный номер этой ячейки
        int count = 1;                 // учитываем диагональный элемент

        // 3) 6 соседей по x,y,z
        const int off[6][3] = {
                { 1,  0,  0}, {-1,  0,  0},
                { 0,  1,  0}, { 0, -1,  0},
                { 0,  0,  1}, { 0,  0, -1}
        };

        for (int n = 0; n < 6; ++n) {
            int ni = i + off[n][0];
            int nj = j + off[n][1];
            int nk = k + off[n][2];
            // проверяем границы
            if (ni < 0 || ni >= W ||
                nj < 0 || nj >= H ||
                nk < 0 || nk >= D) continue;

            int nidx = ni + nj * W + nk * (W * H);

            //  -- если сосед FLUID и fluidNumbers[nidx] > row,
            //     значит мы храним только “верхний треугольник” (симметрично)
            if (labels[nidx] == Utility::FLUID) {
                int nrow = fluidNumbers[nidx];
                if (nrow > row) {
                    count++;
                }
            }
            //  -- если сосед не SOLID (то есть FLUID или AIR),
            //     но мы всё равно добавляем вклад в диагональ,
            //     но не добавляем off-diagonal
            //     (поскольку AIR не даёт off-diagonal,
            //      а SOLID вообще не считается).
            //    Это справедливо, потому что
            //    для AIR → только диагональный вклад,
            //    а off-diagonal (сосед) не появляется в CSR.
        }

        nnz_per_row[row] = count;
    }
};

struct MatrixFiller3D {
    int W, H, D;
    const int* labels;
    const int* fluidNumbers;
    const int* csr_offsets;
    float* csr_values;
    int* csr_columns;
    float scale;

    __device__ void operator()(int idx) const {
        int i = idx % W;
        int j = (idx / W) % H;
        int k = idx / (W * H);

        if (labels[idx] != Utility::FLUID) return;

        int row = fluidNumbers[idx];
        int pos = csr_offsets[row];
        float diagVal = 0.0f;

        // Диагональный элемент
        csr_values[pos] = diagVal;
        csr_columns[pos] = row;
        pos++;

        // Обработка соседей
        const int offsets[6][3] = {
                {1,0,0}, {-1,0,0},
                {0,1,0}, {0,-1,0},
                {0,0,1}, {0,0,-1}
        };

        for (int n = 0; n < 6; n++) {
            int ni = i + offsets[n][0];
            int nj = j + offsets[n][1];
            int nk = k + offsets[n][2];

            if (ni >= 0 && ni < W && nj >= 0 && nj < H && nk >= 0 && nk < D) {
                int nidx = ni + nj * W + nk * W * H;

                if (labels[nidx] == Utility::FLUID) {
                    int col = fluidNumbers[nidx];

                    // Только верхний треугольник
                    if (col > row) {
                        csr_values[pos] = -scale;
                        csr_columns[pos] = col;
                        pos++;
                    }
                    diagVal += scale;
                }
                else if (labels[nidx] == Utility::AIR) {
                    diagVal += scale;
                }
            }
        }

        // Обновляем диагональ
        csr_values[csr_offsets[row]] = diagVal;
    }
};

void FluidSolver3D::constructA(
        thrust::device_vector<float>& csr_values,
        thrust::device_vector<int>& csr_columns,
        thrust::device_vector<int>& csr_offsets,
        thrust::device_vector<int> fluidNumbers
) {
    thrust::device_vector<int> nnz_per_row(fluidCellsAmount, 0);

    // Фаза 1: Подсчет ненулевых элементов
    thrust::for_each_n(
            thrust::device,
            thrust::counting_iterator<int>(0),
            gridWidth * gridHeight * gridDepth,
            MatrixBuilder3D{
                    gridWidth, gridHeight, gridDepth,
                    thrust::raw_pointer_cast(labels.device_ptr()),
                    thrust::raw_pointer_cast(fluidNumbers.data()),
                    thrust::raw_pointer_cast(nnz_per_row.data())
            }
    );

    // Строим смещения
    thrust::exclusive_scan(
            thrust::device,
            nnz_per_row.begin(), nnz_per_row.end(),
            csr_offsets.begin()
    );

    // Общее количество ненулевых элементов
    int total_nnz = thrust::reduce(
            thrust::device,
            nnz_per_row.begin(), nnz_per_row.end()
    );

    csr_values.resize(total_nnz);
    csr_columns.resize(total_nnz);

    // Фаза 2: Заполнение матрицы
    thrust::for_each_n(
            thrust::device,
            thrust::counting_iterator<int>(0),
            gridWidth * gridHeight * gridDepth,
            MatrixFiller3D{
                    gridWidth, gridHeight, gridDepth,
                    thrust::raw_pointer_cast(labels.device_ptr()),
                    thrust::raw_pointer_cast(fluidNumbers.data()),
                    thrust::raw_pointer_cast(csr_offsets.data()),
                    thrust::raw_pointer_cast(csr_values.data()),
                    thrust::raw_pointer_cast(csr_columns.data()),
                    1.0f
            }
    );

    csr_offsets.back() = csr_values.size();

    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "ConstructA error: " << hipGetErrorString(err) << std::endl;
    }
}


struct GlobalToLocal {
    const int* flags;       // Указатель на вектор флагов
    const int* old_to_new;  // Указатель на вектор преобразования индексов
    const float* local_vals; // Указатель на локальные значения

    GlobalToLocal(const int* f, const int* m, const float* l)
            : flags(f), old_to_new(m), local_vals(l) {}

    __host__ __device__
    float operator()(int global_idx) const {
        if (flags[global_idx] == 1) {
            int local_idx = old_to_new[global_idx];
            return local_vals[local_idx];
        }
        return 0.0f; // Значение по умолчанию
    }
};

// Функтор для пометки не-FLUID ячеек (flags[idx]==0) значением −1 в fluidNumbers_d
struct MarkNonFluidFunctor {
    const int* flags;      // указатель на массив flags
    int*       fluidNums;  // указатель на массив fluidNumbers_d

    MarkNonFluidFunctor(const int* _flags, int* _fluidNums)
            : flags(_flags), fluidNums(_fluidNums) {}

    __host__ __device__
    void operator()(int idx) const {
        if (flags[idx] == 0) {
            fluidNums[idx] = -1;
        }
    }
};


int FluidSolver3D::pressureSolve() {

    //w_x_h_x_d = gridWidth * gridHeight * gridDepth;

    // новая нумерация
    thrust::device_vector<int> fluidNumbers_d(w_x_h_x_d, -1);
    thrust::sequence(thrust::device, fluidNumbers_d.begin(), fluidNumbers_d.end()); // последовательность индексов от 0
    thrust::device_vector<int> flags(w_x_h_x_d, 0);

    /*ЧТО ХОТИМ СДЕЛАТЬ НИЖЕ: ВВЕСТИ НОВУЮ НУМЕРАЦИЮ.
     * Ячейка	Метка	flags	fluidNumbers_d
       (0,0,0)	FLUID	 1	     0
       (1,0,0)	SOLID	 0	     1
       (2,0,0)	FLUID	 1	     1
     * */
    thrust::transform(
            thrust::device,
            labels.device_ptr(),
            labels.device_ptr() + w_x_h_x_d,
            flags.begin(),
            FluidFlagFunctor()
    );

//    std::cout << "flags:" << std::endl;
//    thrust::host_vector<float> flags_h = flags;
//    for(int j = 0; j < gridHeight; ++j){
//        for(int i = 0; i < gridWidth; ++i){
//            std::cout << flags_h[i + j*gridWidth] << ", ";
//        }
//        std::cout << std::endl;
//    }


    // с помощью префиксной суммы (не включая текущий жлемент, exclusive) получаем индексы жидких ячеек в новой нумерации (флаги нужны для получения таких сумм)
    thrust::exclusive_scan(
            thrust::device,
            flags.begin(), flags.end(),
            fluidNumbers_d.begin()
    );
    // После этого нужно “записать -1” для тех клеток, где flags[idx]==0:
//    thrust::for_each(
//            thrust::device,
//            thrust::make_counting_iterator<int>(0),
//            thrust::make_counting_iterator<int>(w_x_h_x_d),
//            MarkNonFluidFunctor(
//                    thrust::raw_pointer_cast(flags.data()),
//                    thrust::raw_pointer_cast(fluidNumbers_d.data())
//            )
//    );
    //std::cout << "last number = " << fluidNumbers_d[w_x_h_x_d-1] << std::endl;
    //fluidCellsAmount = fluidNumbers_d[w_x_h_x_d-1];
    //  Подсчет количества жидких ячеек
    fluidCellsAmount = thrust::reduce(
            thrust::device,
            flags.begin(),
            flags.end(),
            0,
            thrust::plus<int>()
    );
//    std::cout << "fluidCelssAmount = " << fluidCellsAmount << std::endl;
    if (fluidCellsAmount == 0) {
        std::cerr << "No fluid cells found!" << std::endl;
        return -1;
    }

    //  Построение правой части (RHS)
    thrust::device_vector<float> rhs_d;
    constructRHS(rhs_d, fluidNumbers_d, flags);

//    std::cout << "fluidNumbers:" << std::endl;
//    thrust::host_vector<float> fluidNumbers = fluidNumbers_d;
//    for(int j = 0; j < gridHeight; ++j){
//        for(int i = 0; i < gridWidth; ++i){
//            std::cout << fluidNumbers[i + j*gridWidth] << ", ";
//        }
//        std::cout << std::endl;
//    }
//
//    std::cout << "rhs for new system:" << std::endl;
//    thrust::host_vector<float> rhs = rhs_d;
//    for(int j = 0; j < gridHeight; ++j){
//        for(int i = 0; i < gridWidth; ++i){
//            if(i + j*gridWidth < rhs.size())
//                std::cout << rhs[i + j*gridWidth] << ", ";
//        }
//        std::cout << std::endl;
//    }

    //  Построение матрицы A в формате CSR
    thrust::device_vector<float> csr_values;
    thrust::device_vector<int> csr_columns;
    thrust::device_vector<int> csr_offsets(fluidCellsAmount + 1, 0);

    constructA(csr_values, csr_columns, csr_offsets, fluidNumbers_d);

    if (csr_values.empty()) {
        std::cerr << "Matrix construction failed!" << std::endl;
        return -2;
    }

    hipStream_t stream = NULL;
    hipStreamCreate(&stream);
    cudssHandle_t handle;
    cudssStatus_t status = cudssCreate(&handle);
    cudssSetStream(handle, stream);

    cudssConfig_t solverConfig;
    cudssData_t solverData;
    cudssConfigCreate(&solverConfig);
    cudssDataCreate(handle, &solverData);

    if (status != CUDSS_STATUS_SUCCESS) {
        std::cerr << "cuDSS init failed: " << status << std::endl;
        return -3;
    }

    cudssMatrix_t A;
    cudssMatrixType_t mtype = CUDSS_MTYPE_SPD;// Symmetric Positive Definite
    cudssMatrixViewType_t mview = CUDSS_MVIEW_UPPER;// Upper triangular stored
    cudssIndexBase_t base = CUDSS_BASE_ZERO;
    int nnz = csr_values.size();

    status = cudssMatrixCreateCsr(
            &A,
            fluidCellsAmount, fluidCellsAmount, nnz,
            thrust::raw_pointer_cast(csr_offsets.data()),
            NULL,
            thrust::raw_pointer_cast(csr_columns.data()),
            thrust::raw_pointer_cast(csr_values.data()),
            HIP_R_32I, HIP_R_32F,
            mtype, mview, base
    );

    if (status != CUDSS_STATUS_SUCCESS) {
        std::cerr << "Matrix creation failed: " << status << std::endl;
        cudssDestroy(handle);
        return -4;
    }

//    thrust::host_vector<float> csr_vals_h = csr_values;
//    thrust::host_vector<float> csr_cols_h = csr_columns;
//    thrust::host_vector<float> csr_offs_h = csr_offsets;

//    for(int i = 0; i < csr_vals_h.size(); ++i){
//        std::cout << csr_vals_h[i] << ", ";
//    }
//    std::cout << "\n------\n";
//    for(int i = 0; i < csr_cols_h.size(); ++i){
//        std::cout << csr_cols_h[i] << ", ";
//    }
//    std::cout << "\n------\n";
//    for(int i = 0; i < csr_offs_h.size(); ++i){
//        std::cout << csr_offs_h[i] << ", ";
//    }
//    std::cout << "\n------\n";
//    std::cin.get();

    // решение Системы линейных алгебраических уравнений с разреженной матрицей
    thrust::device_vector<float> solution(fluidCellsAmount);

    cudssMatrix_t x, b;

    status = cudssMatrixCreateDn(
            &b, fluidCellsAmount, 1, fluidCellsAmount,
            thrust::raw_pointer_cast(rhs_d.data()),
            HIP_R_32F, CUDSS_LAYOUT_COL_MAJOR
    );

    status = cudssMatrixCreateDn(
            &x, fluidCellsAmount, 1, fluidCellsAmount,
            thrust::raw_pointer_cast(solution.data()),
            HIP_R_32F, CUDSS_LAYOUT_COL_MAJOR
    );

    // Решение системы
    // Анализ (async func)
    status = cudssExecute(handle, CUDSS_PHASE_ANALYSIS,
                          solverConfig, solverData, A, x, b);
    hipStreamSynchronize(stream);
    // Факторизация (async func)
    status = cudssExecute(handle, CUDSS_PHASE_FACTORIZATION,
                          solverConfig, solverData, A, x, b);
    hipStreamSynchronize(stream);
    // Решение (async func)
    status = cudssExecute(handle, CUDSS_PHASE_SOLVE,
                          solverConfig, solverData, A, x, b);
    hipStreamSynchronize(stream);

    status = cudssMatrixDestroy(A);
    status = cudssMatrixDestroy(b);
    status = cudssMatrixDestroy(x);
    cudssDataDestroy(handle, solverData);
    cudssConfigDestroy(solverConfig);
    cudssDestroy(handle);
    

//    std::cout << "----solution local---" << std::endl;
//    thrust::host_vector<float> sol_h = solution;
//    for(int k = 0; k < sol_h.size(); ++k){
//        std::cout << sol_h[k] << ", ";
//    }
//    std::cout << std::endl;

    //  Копирование решения в сетку давления
    thrust::device_vector<float> p_temp(w_x_h_x_d, 0.0f);
//    thrust::transform(
//            thrust::device,
//            fluidNumbers_d.begin(), fluidNumbers_d.end(),
//            solution.begin(),
//            p_temp.begin(),
//            CopySolutionFunctor()
//    );
    GlobalToLocal transformer(
            thrust::raw_pointer_cast(flags.data()),
            thrust::raw_pointer_cast(fluidNumbers_d.data()),
            thrust::raw_pointer_cast(solution.data())
    );
    thrust::transform(
            thrust::make_counting_iterator(0),      // Итератор глобальных индексов: 0,1,2,...
            thrust::make_counting_iterator(w_x_h_x_d), // Конец индексов
            p_temp.begin(),                   // Выходной итератор
            transformer                             // Функтор преобразования
    );

    thrust::copy(p_temp.begin(), p_temp.end(), p.device_data.begin());


//    std::cout << "----pressure 3d---" << std::endl;
//    p.host_data = p.device_data;
//    for(int k = 0; k < gridDepth; ++k){
//        for(int j = 0; j < gridHeight; ++j){
//            for(int i = 0; i < gridWidth; ++i){
//                std::cout << p.host_data[i + j*gridWidth + k * gridWidth*gridHeight] << ", ";
//            }
//            std::cout << std::endl;
//        }
//        std::cout << std::endl;
//    }


    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "PressureSolve() error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    return 0;
}

// ----------------------------------
// Утилитарная функция для преобразования из 3D-индекса (i,j,k) в линейный (cell-centered)
__device__ __host__
inline int cellIdx(int i, int j, int k, int W, int H) {
    // Ячейки размером W × H × D
    return i + j * W + k * (W * H);
}

// ----------------------------------
// Функтор для коррекции U-скоростей (MAC-грань по X)
// uSize = (W+1) × H × D
struct UFunctor {
    float*       u;          // массив U-скоростей (face-centered по X), размер (W+1)*H*D
    const float* p;          // массив давлений на центрах ячеек, размер W*H*D
    const int*   labels;     // метки ячеек (AIR, FLUID, SOLID), размер W*H*D
    float        scale;      // = dt / (ρ * dx)
    int          W, H, D;    // размеры сетки (ячейки): W × H × D
    float        VEL_UNKNOWN;

    UFunctor(float*       u_,
             const float* p_,
             const int*   labels_,
             float        scale_,
             int          W_,
             int          H_,
             int          D_,
             float        vel_unknown)
            : u(u_), p(p_), labels(labels_), scale(scale_),
              W(W_), H(H_), D(D_), VEL_UNKNOWN(vel_unknown) {}

    __host__ __device__
    void operator()(int idx) const {
        // Получаем трёхмерные индексы (iU, jU, kU) для u:
        // где iU ∈ [0..W], jU ∈ [0..H-1], kU ∈ [0..D-1]
        int sliceSize = (W + 1) * H;        // размер “плоскости” (iU + jU*(W+1)) для каждого kU
        int kU = idx / sliceSize;
        int rem = idx % sliceSize;
        int jU = rem / (W + 1);
        int iU = rem % (W + 1);

        float usolid = 0.0f; // пока что границы неподвижны
        float invScale = 1.0f / scale;
        float p0 = 0.0f;
        float p1 = 0.0f;
        int leftIdx  = cellIdx(iU - 1, jU, kU, W, H);
        int rightIdx = cellIdx(iU, jU, kU, W, H);
        if(labels[leftIdx] != Utility::SOLID && labels[rightIdx] != Utility::SOLID){
            p0 = p[leftIdx];
            p1 = p[rightIdx];
        } else if(labels[leftIdx] == Utility::SOLID){
            p0 = p[rightIdx] - invScale * (u[idx] - usolid);
            p1 = p[rightIdx];
        } else{
            p0 = p[leftIdx];
            p1 = p[leftIdx] + invScale * (u[idx] - usolid);
        }
        u[idx] = u[idx] - scale * (p1 - p0);
        // Коррекцию u делаем только для внутренних граней: iU от 1 до W-1
        // if (iU > 0 && iU < W) {
        //     // Соответствующие “левые” и “правые” ячейки:
        //     // leftCell  = (iU-1, jU, kU)
        //     // rightCell = (iU,   jU, kU)
        //     int leftIdx  = cellIdx(iU - 1, jU, kU, W, H);
        //     int rightIdx = cellIdx(iU, jU, kU, W, H);

        //     bool leftFluid  = (labels[leftIdx]  == Utility::FLUID);
        //     bool rightFluid = (labels[rightIdx] == Utility::FLUID);
        //     bool leftSolid  = (labels[leftIdx]  == Utility::SOLID);
        //     bool rightSolid = (labels[rightIdx] == Utility::SOLID);

        //     if (leftFluid || rightFluid) {
        //         // Если хотя бы один сосед FLUID, но нет SOLID — корректируем
        //         if (leftSolid || rightSolid) {
        //             // Грань у стены — обнуляем скорость
        //             u[idx] = 0.0f;
        //         } else {
        //             // Коррекция по градиенту: u ← u − scale * (p[right] − p[left])
        //             float dp = p[rightIdx] - p[leftIdx];
        //             u[idx] -= scale * dp;
        //         }
        //         return;
        //     }
        //     return; // внутренняя + слева и справа AIR -> без изменений
        // }
        // // Во всех остальных случаях (грань между AIR или на границе) помечаем неизвестной
        // u[idx] = 0.0f;
    }
};

// ----------------------------------
// Функтор для коррекции V-скоростей (MAC-грань по Y)
// vSize = W × (H+1) × D
struct VFunctor {
    float*       v;
    const float* p;
    const int*   labels;
    float        scale;
    int          W, H, D;
    float        VEL_UNKNOWN;

    VFunctor(float*       v_,
             const float* p_,
             const int*   labels_,
             float        scale_,
             int          W_,
             int          H_,
             int          D_,
             float        vel_unknown)
            : v(v_), p(p_), labels(labels_), scale(scale_),
              W(W_), H(H_), D(D_), VEL_UNKNOWN(vel_unknown) {}

    __host__ __device__
    void operator()(int idx) const {
        // iV ∈ [0..W-1], jV ∈ [0..H], kV ∈ [0..D-1]
        int sliceSize = W * (H + 1);
        int kV = idx / sliceSize;
        int rem = idx % sliceSize;
        int jV = rem / W;
        int iV = rem % W;
        
        float vsolid = 0.0f; // пока что границы неподвижны
        float invScale = 1.0f / scale;
        float p0 = 0.0f;
        float p1 = 0.0f;
        int leftIdx  = cellIdx(iV, jV - 1, kV, W, H);
        int rightIdx = cellIdx(iV, jV, kV, W, H);
        if(labels[leftIdx] != Utility::SOLID && labels[rightIdx] != Utility::SOLID){
            p0 = p[leftIdx];
            p1 = p[rightIdx];
        } else if(labels[leftIdx] == Utility::SOLID){
            p0 = p[rightIdx] - invScale * (v[idx] - vsolid);
            p1 = p[rightIdx];
        } else{
            p0 = p[leftIdx];
            p1 = p[leftIdx] + invScale * (v[idx] - vsolid);
        }
        v[idx] = v[idx] - scale * (p1 - p0);

        // // Коррекция только для внутренних граней: jV от 1 до H-1
        // if (jV > 0 && jV < H) {
        //     // leftCell  = (iV, jV-1, kV)
        //     // rightCell = (iV, jV,   kV)
        //     int leftIdx  = cellIdx(iV, jV - 1, kV, W, H);
        //     int rightIdx = cellIdx(iV, jV, kV, W, H);

        //     bool leftFluid  = (labels[leftIdx]  == Utility::FLUID);
        //     bool rightFluid = (labels[rightIdx] == Utility::FLUID);
        //     bool leftSolid  = (labels[leftIdx]  == Utility::SOLID);
        //     bool rightSolid = (labels[rightIdx] == Utility::SOLID);

        //     if (leftFluid || rightFluid) {
        //         if (leftSolid || rightSolid) {
        //             v[idx] = 0.0f;
        //         } else {
        //             float dp = p[rightIdx] - p[leftIdx];
        //             v[idx] -= scale * dp;
        //         }
        //         return;
        //     }
        //     return;
        // }
        // v[idx] = 0.0f;
    }
};

// ----------------------------------
// Функтор для коррекции W-скоростей (MAC-грань по Z)
// wSize = W × H × (D+1)
struct WFunctor {
    float*       w;
    const float* p;
    const int*   labels;
    float        scale;
    int          W, H, D;
    float        VEL_UNKNOWN;

    WFunctor(float*       w_,
             const float* p_,
             const int*   labels_,
             float        scale_,
             int          W_,
             int          H_,
             int          D_,
             float        vel_unknown)
            : w(w_), p(p_), labels(labels_), scale(scale_),
              W(W_), H(H_), D(D_), VEL_UNKNOWN(vel_unknown) {}

    __host__ __device__
    void operator()(int idx) const {
        // iW ∈ [0..W-1], jW ∈ [0..H-1], kW ∈ [0..D]
        int sliceSize = W * H;
        int kW = idx / sliceSize;
        int rem = idx % sliceSize;
        int jW = rem / W;
        int iW = rem % W;

        float wsolid = 0.0f; // пока что границы неподвижны
        float invScale = 1.0f / scale;
        float p0 = 0.0f;
        float p1 = 0.0f;
        int leftIdx  = cellIdx(iW, jW, kW - 1, W, H);
        int rightIdx = cellIdx(iW, jW, kW, W, H);
        if(labels[leftIdx] != Utility::SOLID && labels[rightIdx] != Utility::SOLID){
            p0 = p[leftIdx];
            p1 = p[rightIdx];
        } else if(labels[leftIdx] == Utility::SOLID){
            p0 = p[rightIdx] - invScale * (w[idx] - wsolid);
            p1 = p[rightIdx];
        } else{
            p0 = p[leftIdx];
            p1 = p[leftIdx] + invScale * (w[idx] - wsolid);
        }
        w[idx] = w[idx] - scale * (p1 - p0);

        // // Коррекция только для внутренних граней: kW от 1 до D-1
        // if (kW > 0 && kW < D) {
        //     // leftCell  = (iW, jW, kW-1)
        //     // rightCell = (iW, jW, kW)
        //     int leftIdx  = cellIdx(iW, jW, kW - 1, W, H);
        //     int rightIdx = cellIdx(iW, jW, kW, W, H);

        //     bool leftFluid  = (labels[leftIdx]  == Utility::FLUID);
        //     bool rightFluid = (labels[rightIdx] == Utility::FLUID);
        //     bool leftSolid  = (labels[leftIdx]  == Utility::SOLID);
        //     bool rightSolid = (labels[rightIdx] == Utility::SOLID);

        //     if (leftFluid || rightFluid) {
        //         if (leftSolid || rightSolid) {
        //             w[idx] = 0.0f;
        //         } else {
        //             float dp = p[rightIdx] - p[leftIdx];
        //             w[idx] -= scale * dp;
        //         }
        //         return;
        //     }
        //     return;
        // }
        // w[idx] = 0.0f;
    }
};

// ----------------------------------
// 4) Обновлённый метод FluidSolver3D::applyPressure()
void FluidSolver3D::applyPressure() {
    float scale        = dt / (FLUID_DENSITY * dx);
    float vel_unknown  = static_cast<float>(VEL_UNKNOWN);

//    std::cout << "----pressure 3d---" << std::endl;
//    p.host_data = p.device_data;
//    for(int k = 0; k < gridDepth; ++k){
//        for(int j = 0; j < gridHeight; ++j){
//            for(int i = 0; i < gridWidth; ++i){
//                std::cout << p.host_data[i + j*gridWidth + k * gridWidth*gridHeight] << ", ";
//            }
//            std::cout << std::endl;
//        }
//        std::cout <<"\n"<< std::endl;
//    }

    // --- Коррекция U-скоростей ---
    int u_size = (gridWidth + 1) * gridHeight * gridDepth;
    thrust::for_each(
            thrust::device,
            thrust::make_counting_iterator(0),
            thrust::make_counting_iterator(u_size),
            UFunctor(
                    u.device_ptr(),
                    p.device_ptr(),
                    labels.device_ptr(),
                    scale,
                    gridWidth,
                    gridHeight,
                    gridDepth,
                    vel_unknown
            )
    );

//        u.copy_to_host();
//        std::cout << "----u 3d (after pressure apply)---" << std::endl;
//    for(int k = 0; k < gridDepth; ++k){
//        for(int j = 0; j < gridHeight; ++j){
//            for(int i = 0; i < gridWidth+1; ++i){
//                std::cout << u.host_data[i + j*(gridWidth+1) + k * (gridWidth+1)*gridHeight] << ", ";
//            }
//            std::cout << std::endl;
//        }
//        std::cout << std::endl;
//    }


    // --- Коррекция V-скоростей ---
    int v_size = gridWidth * (gridHeight + 1) * gridDepth;
    thrust::for_each(
            thrust::device,
            thrust::make_counting_iterator(0),
            thrust::make_counting_iterator(v_size),
            VFunctor(
                    v.device_ptr(),
                    p.device_ptr(),
                    labels.device_ptr(),
                    scale,
                    gridWidth,
                    gridHeight,
                    gridDepth,
                    vel_unknown
            )
    );

//        v.copy_to_host();
//    std::cout << "----v 3d (after pressure apply)---" << std::endl;
//    for(int k = 0; k < gridDepth; ++k){
//        for(int j = 0; j < gridHeight+1; ++j){
//            for(int i = 0; i < gridWidth; ++i){
//                std::cout << v.host_data[i + j*(gridWidth) + k * (gridWidth)*(gridHeight+1)] << ", ";
//            }
//            std::cout << std::endl;
//        }
//        std::cout << std::endl;
//    }

    // --- Коррекция W-скоростей ---
    int w_size = gridWidth * gridHeight * (gridDepth + 1);
    thrust::for_each(
            thrust::device,
            thrust::make_counting_iterator(0),
            thrust::make_counting_iterator(w_size),
            WFunctor(
                    w.device_ptr(),
                    p.device_ptr(),
                    labels.device_ptr(),
                    scale,
                    gridWidth,
                    gridHeight,
                    gridDepth,
                    vel_unknown
            )
    );

//        w.copy_to_host();
//    std::cout << "----w 3d (after pressure apply)---" << std::endl;
//    for(int k = 0; k < gridDepth+1; ++k){
//        for(int j = 0; j < gridHeight; ++j){
//            for(int i = 0; i < gridWidth; ++i){
//                std::cout << w.host_data[i + j*(gridWidth) + k * (gridWidth)*(gridHeight)] << ", ";
//            }
//            std::cout << std::endl;
//        }
//        std::cout << std::endl;
//    }

    // Проверка ошибок CUDA
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "applyPressure() error: " << hipGetErrorString(err) << std::endl;
    }
}

__host__ void FluidSolver3D::frameStep(){
    labelGrid();

    //particles velocities to grid
    particlesToGrid();

    //saving a copy of the current grid velocities (for FLIP)
    saveVelocities();

    //applying body forces on grid (e.g. gravity force)
    applyForces();
    pressureSolve();
    applyPressure();
    //grid velocities to particles
    gridToParticles(PIC_WEIGHT);

    advectParticles(ADVECT_MAX);

}

__host__ void FluidSolver3D::run(int max_steps) {
    d_particles = h_particles;
    // Prepare
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Start record
    hipEventRecord(start, 0);
    for(int i = 0; i < max_steps; ++i){
        frameStep();
        if(i%10 == 0){
            h_particles = d_particles;
            Utility::save3dParticlesToPLY(h_particles, "InputData/particles_" + std::to_string(i) + ".ply");
            std::cout << "frame = " << i/10 << "; numParticles = " << h_particles.size()<<std::endl;
        }

    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop); // that's our time!
    std::cout << "elapsed time = " << elapsedTime / 1000.0f << std::endl;
}
