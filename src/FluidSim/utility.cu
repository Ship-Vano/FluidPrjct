#include "hip/hip_runtime.h"
#include "utility.cuh"

__host__ __device__ float3 operator+(const float3& a, const float3& b) {
    return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__host__ __device__ float3 operator-(const float3& a, const float3& b) {
    return make_float3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__host__ __device__ float3 operator*(const float3& a, float b) {
    return make_float3(a.x * b, a.y * b, a.z * b);
}

__host__ __device__ float3 operator*(float b, const float3& a){
    return a * b;
}

__host__ __device__ float operator*(const float3& a, const float3& b){
    return a.x*b.x + a.y*b.y + a.z*b.z;
}

__host__ __device__ float3 operator/(const float3&a, const float&b){
    return a * (1.0f/b);
}

__host__ __device__ double3 operator+(const double3& a, const double3& b) {
    return make_double3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__host__ __device__ double3 operator-(const double3& a, const double3& b) {
    return make_double3(a.x - b.x, a.y - b.y, a.z - b.z);
}

__device__ double3 operator*(const double3& a, double b) {
    return make_double3(a.x * b, a.y * b, a.z * b);
}

__device__ float2 operator+(const float2& a, const float2& b) {
    return make_float2(a.x + b.x, a.y + b.y);
}

__device__ float2 operator-(const float2& a, const float2& b) {
    return make_float2(a.x - b.x, a.y - b.y);
}

__device__ float2 operator*(const float2& a, float b) {
    return make_float2(a.x * b, a.y * b);
}



namespace Utility {
    float2 getGridCellPosition(float i, float j, float dx) {
        return make_float2((i +0.5f)* dx, (j + 0.5f) * dx);
    }

    float3 getGridCellPosition(float i, float j, float k, float dx) {
        return make_float3((i +0.5f)* dx, (j + 0.5f) * dx, (k + 0.5f) * dx);
    }

    void saveParticlesToFile(const std::vector<Particle2D>& particles,
                             const std::string& filename){
        std::ofstream file(filename, std::ios::binary);

        // Просто записываем x и y как два отдельных float
        for (const auto& p : particles) {
            float x = p.pos.x;
            float y = p.pos.y;
            file.write(reinterpret_cast<const char*>(&x), sizeof(float));
            file.write(reinterpret_cast<const char*>(&y), sizeof(float));
        }
    }
    void saveParticlesToPLY(const std::vector<Particle2D>& particles,
                            const std::string& filename) {
        std::ofstream file(filename);

        file << "ply\n"
             << "format ascii 1.0\n"
             << "element vertex " << particles.size() << "\n"
             << "property float x\n"
             << "property float y\n"
             << "property float z\n"
             << "end_header\n";

        for (const auto p: particles) {
            file << p.pos.x << " " << p.pos.y << " " << 0.0 << "\n";
        }
    }

    void save3dParticlesToPLY(const thrust::host_vector<Particle3D>& particles,
                              const std::string& filename) {
        std::ofstream file(filename);
        if (!file.is_open()) {
            std::cerr << "Error opening file: " << filename << std::endl;
            return;
        }
        file << "ply\n"
             << "format ascii 1.0\n"
             << "element vertex " << particles.size() << "\n"
             << "property float x\n"
             << "property float y\n"
             << "property float z\n"
             << "end_header\n";

        for (const auto p: particles) {
            file << p.pos.x << " " << p.pos.y << " " << p.pos.z << "\n";
        }
    }

    void save3dParticlesToOFF(const thrust::host_vector<Particle3D>& particles,
                              const std::string& filename) {
        std::ofstream file(filename);
        if (!file.is_open()) {
            std::cerr << "Error opening file: " << filename << std::endl;
            return;
        }

        // Заголовок формата OFF
        file << "OFF\n";
        file << particles.size() << " 0 0\n"; // Вершины, грани (0), рёбра (0)

        // Запись координат частиц
        for (const auto& p : particles) {
            file << p.pos.x << " " << p.pos.y << " " << p.pos.z << "\n";
        }
    }

    int2 getGridCellIndex(float2 pos, float dx){
        return make_int2((int)(pos.x / dx), (int)(pos.y/dx));
    }

    __device__ int getGridCellIndex_device(float2 pos, float dx, int gridWidth){
        return (int)(pos.x / dx)  + (int)(pos.y / dx) * gridWidth;
    }


    __device__ float2 getGridCellPosition_device(float i, float j, float dx){
        return float2{(i+0.5f)*dx, (j+0.5f)*dx};
    }

    __device__ float bilinearHatKernel(float2 dist, float dx, float dy){
        return hatFunction(dist.x / dx) * hatFunction(dist.y / dy);
    }

    __device__ float hatFunction(float r){
        float rAbs = fabs(r);
        if(rAbs-1.0f <= 1e-8){ //if(rAbs <= 1.0)
            return 1.0f - rAbs;
        } else{
            return 0.0f;
        }
    }

    __device__ int2 getGridIndicesU(int ind, int gridWidth) {
        return {ind % (gridWidth+1), ind / (gridWidth+1)}; // i, j для u-компоненты
    }

    __device__ int2 getGridIndicesV(int ind, int gridWidth) {
        return {ind % (gridWidth), ind / (gridWidth)}; // i, j для v-компоненты
    }

    __device__ bool contains(float* sdf_data, float3 sdf_origin, float3 body_pos, float3 local_com, float* rotation_matrix, float3 world_pos, float sdf_cell_size, int sdf_w, int sdf_h, int sdf_d){
        // 1. Преобразование в локальные координаты тела
        float3 p_rel = world_pos - body_pos;

        // 2. Обратное вращение: p_local_centered = R^T * p_rel
        float3 p_local_centered;
        p_local_centered.x = rotation_matrix[0] * p_rel.x
                           + rotation_matrix[3] * p_rel.y
                           + rotation_matrix[6] * p_rel.z;

        p_local_centered.y = rotation_matrix[1] * p_rel.x
                           + rotation_matrix[4] * p_rel.y
                           + rotation_matrix[7] * p_rel.z;

        p_local_centered.z = rotation_matrix[2] * p_rel.x
                           + rotation_matrix[5] * p_rel.y
                           + rotation_matrix[8] * p_rel.z;

        // 3. Переход в SDF-систему: добавляем локальный центр масс
        float3 p_local = p_local_centered + local_com;

        // 4. Преобразование в координаты SDF сетки
        float3 sdf_coord = {
            (p_local.x - sdf_origin.x) / sdf_cell_size,
            (p_local.y - sdf_origin.y) / sdf_cell_size,
            (p_local.z - sdf_origin.z) / sdf_cell_size
        };
        // 5. Проверка границ сетки
        if (sdf_coord.x < 0 || sdf_coord.x >= sdf_w ||
            sdf_coord.y < 0 || sdf_coord.y >= sdf_h ||
            sdf_coord.z < 0 || sdf_coord.z >= sdf_d)
        {
            return false;
        }

        // 6. Определение индексов
        int i = min(static_cast<int>(sdf_coord.x), sdf_w - 1);
        int j = min(static_cast<int>(sdf_coord.y), sdf_h - 1);
        int k = min(static_cast<int>(sdf_coord.z), sdf_d - 1);

        // 7. Проверка значения SDF
        int idx = i + j * sdf_w + k * sdf_w * sdf_h;
        return sdf_data[idx] <= 0.0f;
    }

    __host__ __device__ float3 cross(const float3& a, const float3& b){
        return make_float3(
                a.y * b.z - a.z * b.y,
                a.z * b.x - a.x * b.z,
                a.x * b.y - a.y * b.x

        );
    }

    __device__ float sampleBody(float3 bodyVel, float3 bodyOmega, float3 bodyCM, float3 facePos, float3 normal){
        float3 vel = bodyVel + cross(bodyOmega, facePos - bodyCM);
        return vel * normal;
    }

    __host__ __device__ float4 multiply_quaternions(const float4& a, const float4& b) {
        return {
                a.w*b.x + a.x*b.w + a.y*b.z - a.z*b.y,
                a.w*b.y - a.x*b.z + a.y*b.w + a.z*b.x,
                a.w*b.z + a.x*b.y - a.y*b.x + a.z*b.w,
                a.w*b.w - a.x*b.x - a.y*b.y - a.z*b.z
        };
    }

    // Нормализация кватерниона
    __host__ __device__ void normalize_quaternion(float4& q) {
        float len = sqrt(q.x*q.x + q.y*q.y + q.z*q.z + q.w*q.w);
        if (len > 1e-6f) {
            q.x /= len;
            q.y /= len;
            q.z /= len;
            q.w /= len;
        }
    }

    // Преобразование кватерниона в корабельные углы
    __host__ __device__ float3 quaternion_to_ship_angles(const float4& q) {
        // Крен (roll, φ) - вращение вокруг оси X
        // Тангаж (pitch, θ) - вращение вокруг оси Y
        // Рыскание (yaw, ψ) - вращение вокруг оси Z

        // Используем формулу для преобразования кватерниона в углы Эйлера (Z-Y-X)
        float roll, pitch, yaw;

        // Рыскание (yaw) - ψ
        float siny_cosp = 2.0f * (q.w * q.z + q.x * q.y);
        float cosy_cosp = 1.0f - 2.0f * (q.y * q.y + q.z * q.z);
        yaw = atan2(siny_cosp, cosy_cosp);

        // Тангаж (pitch) - θ
        float sinp = 2.0f * (q.w * q.y - q.z * q.x);
        if (fabs(sinp) >= 1.0f) {
            // Используем 90 градусов, если значение выходит за пределы
            pitch = copysign(M_PI / 2.0f, sinp);
        } else {
            pitch = asin(sinp);
        }

        // Крен (roll) - φ
        float sinr_cosp = 2.0f * (q.w * q.x + q.y * q.z);
        float cosr_cosp = 1.0f - 2.0f * (q.x * q.x + q.y * q.y);
        roll = atan2(sinr_cosp, cosr_cosp);

        return make_float3(roll, pitch, yaw);
    }

}