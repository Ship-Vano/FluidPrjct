#include "hip/hip_runtime.h"
#include "test.cuh"

__device__ float3 calc(){
    float3 vec1{1.0f, 2.0f, 3.0f};
    float3 vec2{1.0f, 2.0f, 3.0f};
    float3 vec3 = vec1 + vec2;
    return vec3;
}

__global__ void run_calc_kernel(float3* result) {
    *result = calc();  // Вызов device-функции внутри kernel
}

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << msg << " - " << hipGetErrorString(err) << std::endl;
        exit(1);
    }
}


 void out() {
     float3* d_result;  // Указатель на память GPU
     hipError_t err = hipMalloc(&d_result, sizeof(float3));
     checkCudaError(err, "hipMalloc failed");

     // Запуск kernel (1 блок, 1 поток)
     run_calc_kernel<<<1, 1>>>(d_result);

     float3 vec;
     hipMemcpy(&vec, d_result, sizeof(float3), hipMemcpyDeviceToHost);

     std::cout << "Vec: (" << vec.x << ", " << vec.y << ", " << vec.z << ")" << std::endl;
     hipFree(d_result);

    std::cout << "Hello, World!" << std::endl;


     // 1. Инициализация cuSPARSE
     hipsparseHandle_t handle;
     hipsparseCreate(&handle);

     // 2. Данные матрицы (CSR-формат)
     int n = 3;  // Размер матрицы 3x3
     float h_values[] = {1.0f, 2.0f, 3.0f};  // Ненулевые элементы
     int h_col_ind[] = {0, 1, 2};            // Столбцы
     int h_row_ptr[] = {0, 1, 2, 3};         // Индексы строк

     // 3. Копируем данные на GPU
     float* d_values;
     int* d_col_ind, *d_row_ptr;
     hipMalloc(&d_values, 3 * sizeof(float));
     hipMalloc(&d_col_ind, 3 * sizeof(int));
     hipMalloc(&d_row_ptr, 4 * sizeof(int));

     hipMemcpy(d_values, h_values, 3 * sizeof(float), hipMemcpyHostToDevice);
     hipMemcpy(d_col_ind, h_col_ind, 3 * sizeof(int), hipMemcpyHostToDevice);
     hipMemcpy(d_row_ptr, h_row_ptr, 4 * sizeof(int), hipMemcpyHostToDevice);

     // 4. Создаем дескриптор матрицы
     hipsparseSpMatDescr_t matA;
     hipsparseCreateCsr(&matA,
                       n, n, 3,  // rows, cols, nnz
                       d_row_ptr, d_col_ind, d_values,
                       HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                       HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F);

     // 5. Создаем дескрипторы векторов
     float* d_x, *d_y;
     hipMalloc(&d_x, n * sizeof(float));
     hipMalloc(&d_y, n * sizeof(float));

     float h_x[] = {1.0f, 1.0f, 1.0f};  // Входной вектор
     hipMemcpy(d_x, h_x, n * sizeof(float), hipMemcpyHostToDevice);

     hipsparseDnVecDescr_t vecX, vecY;
     hipsparseCreateDnVec(&vecX, n, d_x, HIP_R_32F);
     hipsparseCreateDnVec(&vecY, n, d_y, HIP_R_32F);

     // 6. Настройка операции SpMV
     float alpha = 1.0f, beta = 0.0f;
     size_t bufferSize;  // Объявляем переменную для размера буфера
     hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                             &alpha, matA, vecX, &beta, vecY,
                             HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize);

     void* buffer;
     hipMalloc(&buffer, bufferSize);

     // 7. Умножение матрицы на вектор: y = alpha * A * x + beta * y
     hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                  &alpha, matA, vecX, &beta, vecY,
                  HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, buffer);

     // 8. Копируем результат на CPU
     float h_y[3];
     hipMemcpy(h_y, d_y, n * sizeof(float), hipMemcpyDeviceToHost);
     printf("Result: %f, %f, %f\n", h_y[0], h_y[1], h_y[2]);

     // 9. Освобождаем ресурсы
     hipsparseDestroySpMat(matA);
     hipsparseDestroyDnVec(vecX);
     hipsparseDestroyDnVec(vecY);
     hipsparseDestroy(handle);
     hipFree(d_values);
     hipFree(d_col_ind);
     hipFree(d_row_ptr);
     hipFree(d_x);
     hipFree(d_y);
     hipFree(buffer);
}

